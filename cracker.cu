#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <SDL.h>
#include <string.h>
#include <openssl/md5.h>

// CITE CODE:  http://openwall.info/wiki/people/solar/software/public-domain-source-code/md5 AND CHARLIE + MENTORS

#define THREADS_PER_BLOCK 72
#define LENGTH 8
#define NUM_CHAR 36

typedef unsigned int GPU_MD5_u32plus;
 
typedef struct {
  GPU_MD5_u32plus lo, hi;
  GPU_MD5_u32plus a, b, c, d;
  unsigned char buffer[64];
  GPU_MD5_u32plus block[16];
} GPU_MD5_CTX;
 
/*
 * The basic MD5 functions.
 *
 * F and G are optimized compared to their RFC 1321 definitions for
 * architectures that lack an AND-NOT instruction, just like in Colin Plumb's
 * implementation.
 */
#define F(x, y, z)			((z) ^ ((x) & ((y) ^ (z))))
#define G(x, y, z)			((y) ^ ((z) & ((x) ^ (y))))
#define H(x, y, z)			(((x) ^ (y)) ^ (z))
#define H2(x, y, z)			((x) ^ ((y) ^ (z)))
#define I(x, y, z)			((y) ^ ((x) | ~(z)))
 
/*
 * The MD5 transformation for all four rounds.
 */
#define STEP(f, a, b, c, d, x, t, s)                            \
  (a) += f((b), (c), (d)) + (x) + (t);                          \
  (a) = (((a) << (s)) | (((a) & 0xffffffff) >> (32 - (s))));    \
  (a) += (b);
 
/*
 * SET reads 4 input bytes in little-endian byte order and stores them in a
 * properly aligned word in host byte order.
 *
 * The check for little-endian architectures that tolerate unaligned memory
 * accesses is just an optimization.  Nothing will break if it fails to detect
 * a suitable architecture.
 *
 * Unfortunately, this optimization may be a C strict aliasing rules violation
 * if the caller's data buffer has effective type that cannot be aliased by
 * MD5_u32plus.  In practice, this problem may occur if these MD5 routines are
 * inlined into a calling function, or with future and dangerously advanced
 * link-time optimizations.  For the time being, keeping these MD5 routines in
 * their own translation unit avoids the problem.
 */
#if defined(__i386__) || defined(__x86_64__) || defined(__vax__)
#define SET(n)                                  \
  (*(GPU_MD5_u32plus *)&ptr[(n) * 4])
#define GET(n)                                  \
  SET(n)
#else
#define SET(n)                                  \
  (ctx->block[(n)] =                            \
   (GPU_MD5_u32plus)ptr[(n) * 4] |              \
   ((GPU_MD5_u32plus)ptr[(n) * 4 + 1] << 8) |   \
   ((GPU_MD5_u32plus)ptr[(n) * 4 + 2] << 16) |  \
   ((GPU_MD5_u32plus)ptr[(n) * 4 + 3] << 24))
#define GET(n)                                  \
  (ctx->block[(n)])
#endif
 
/*
 * This processes one or more 64-byte data blocks, but does NOT update the bit
 * counters.  There are no alignment requirements.
 */
__device__ static const void *body(GPU_MD5_CTX *ctx, const void *data, unsigned long size)
{
  const unsigned char *ptr;
  GPU_MD5_u32plus a, b, c, d;
  GPU_MD5_u32plus saved_a, saved_b, saved_c, saved_d;
 
  ptr = (const unsigned char *)data;
 
  a = ctx->a;
  b = ctx->b;
  c = ctx->c;
  d = ctx->d;
 
  do {
    saved_a = a;
    saved_b = b;
    saved_c = c;
    saved_d = d;
 
    /* Round 1 */
    STEP(F, a, b, c, d, SET(0), 0xd76aa478, 7)
      STEP(F, d, a, b, c, SET(1), 0xe8c7b756, 12)
      STEP(F, c, d, a, b, SET(2), 0x242070db, 17)
      STEP(F, b, c, d, a, SET(3), 0xc1bdceee, 22)
      STEP(F, a, b, c, d, SET(4), 0xf57c0faf, 7)
      STEP(F, d, a, b, c, SET(5), 0x4787c62a, 12)
      STEP(F, c, d, a, b, SET(6), 0xa8304613, 17)
      STEP(F, b, c, d, a, SET(7), 0xfd469501, 22)
      STEP(F, a, b, c, d, SET(8), 0x698098d8, 7)
      STEP(F, d, a, b, c, SET(9), 0x8b44f7af, 12)
      STEP(F, c, d, a, b, SET(10), 0xffff5bb1, 17)
      STEP(F, b, c, d, a, SET(11), 0x895cd7be, 22)
      STEP(F, a, b, c, d, SET(12), 0x6b901122, 7)
      STEP(F, d, a, b, c, SET(13), 0xfd987193, 12)
      STEP(F, c, d, a, b, SET(14), 0xa679438e, 17)
      STEP(F, b, c, d, a, SET(15), 0x49b40821, 22)
 
      /* Round 2 */
      STEP(G, a, b, c, d, GET(1), 0xf61e2562, 5)
      STEP(G, d, a, b, c, GET(6), 0xc040b340, 9)
      STEP(G, c, d, a, b, GET(11), 0x265e5a51, 14)
      STEP(G, b, c, d, a, GET(0), 0xe9b6c7aa, 20)
      STEP(G, a, b, c, d, GET(5), 0xd62f105d, 5)
      STEP(G, d, a, b, c, GET(10), 0x02441453, 9)
      STEP(G, c, d, a, b, GET(15), 0xd8a1e681, 14)
      STEP(G, b, c, d, a, GET(4), 0xe7d3fbc8, 20)
      STEP(G, a, b, c, d, GET(9), 0x21e1cde6, 5)
      STEP(G, d, a, b, c, GET(14), 0xc33707d6, 9)
      STEP(G, c, d, a, b, GET(3), 0xf4d50d87, 14)
      STEP(G, b, c, d, a, GET(8), 0x455a14ed, 20)
      STEP(G, a, b, c, d, GET(13), 0xa9e3e905, 5)
      STEP(G, d, a, b, c, GET(2), 0xfcefa3f8, 9)
      STEP(G, c, d, a, b, GET(7), 0x676f02d9, 14)
      STEP(G, b, c, d, a, GET(12), 0x8d2a4c8a, 20)
 
      /* Round 3 */
      STEP(H, a, b, c, d, GET(5), 0xfffa3942, 4)
      STEP(H2, d, a, b, c, GET(8), 0x8771f681, 11)
      STEP(H, c, d, a, b, GET(11), 0x6d9d6122, 16)
      STEP(H2, b, c, d, a, GET(14), 0xfde5380c, 23)
      STEP(H, a, b, c, d, GET(1), 0xa4beea44, 4)
      STEP(H2, d, a, b, c, GET(4), 0x4bdecfa9, 11)
      STEP(H, c, d, a, b, GET(7), 0xf6bb4b60, 16)
      STEP(H2, b, c, d, a, GET(10), 0xbebfbc70, 23)
      STEP(H, a, b, c, d, GET(13), 0x289b7ec6, 4)
      STEP(H2, d, a, b, c, GET(0), 0xeaa127fa, 11)
      STEP(H, c, d, a, b, GET(3), 0xd4ef3085, 16)
      STEP(H2, b, c, d, a, GET(6), 0x04881d05, 23)
      STEP(H, a, b, c, d, GET(9), 0xd9d4d039, 4)
      STEP(H2, d, a, b, c, GET(12), 0xe6db99e5, 11)
      STEP(H, c, d, a, b, GET(15), 0x1fa27cf8, 16)
      STEP(H2, b, c, d, a, GET(2), 0xc4ac5665, 23)
 
      /* Round 4 */
      STEP(I, a, b, c, d, GET(0), 0xf4292244, 6)
      STEP(I, d, a, b, c, GET(7), 0x432aff97, 10)
      STEP(I, c, d, a, b, GET(14), 0xab9423a7, 15)
      STEP(I, b, c, d, a, GET(5), 0xfc93a039, 21)
      STEP(I, a, b, c, d, GET(12), 0x655b59c3, 6)
      STEP(I, d, a, b, c, GET(3), 0x8f0ccc92, 10)
      STEP(I, c, d, a, b, GET(10), 0xffeff47d, 15)
      STEP(I, b, c, d, a, GET(1), 0x85845dd1, 21)
      STEP(I, a, b, c, d, GET(8), 0x6fa87e4f, 6)
      STEP(I, d, a, b, c, GET(15), 0xfe2ce6e0, 10)
      STEP(I, c, d, a, b, GET(6), 0xa3014314, 15)
      STEP(I, b, c, d, a, GET(13), 0x4e0811a1, 21)
      STEP(I, a, b, c, d, GET(4), 0xf7537e82, 6)
      STEP(I, d, a, b, c, GET(11), 0xbd3af235, 10)
      STEP(I, c, d, a, b, GET(2), 0x2ad7d2bb, 15)
      STEP(I, b, c, d, a, GET(9), 0xeb86d391, 21)
 
      a += saved_a;
    b += saved_b;
    c += saved_c;
    d += saved_d;
 
    ptr += 64;
  } while (size -= 64);
 
  ctx->a = a;
  ctx->b = b;
  ctx->c = c;
  ctx->d = d;
 
  return ptr;
}
 
__device__ void GPU_MD5_Init(GPU_MD5_CTX *ctx)
{
  ctx->a = 0x67452301;
  ctx->b = 0xefcdab89;
  ctx->c = 0x98badcfe;
  ctx->d = 0x10325476;
 
  ctx->lo = 0;
  ctx->hi = 0;
}
 
__device__ void GPU_MD5_Update(GPU_MD5_CTX *ctx, const void *data, unsigned long size)
{
  GPU_MD5_u32plus saved_lo;
  unsigned long used, available;
 
  saved_lo = ctx->lo;
  if ((ctx->lo = (saved_lo + size) & 0x1fffffff) < saved_lo)
    ctx->hi++;
  ctx->hi += size >> 29;
 
  used = saved_lo & 0x3f;
 
  if (used) {
    available = 64 - used;
 
    if (size < available) {
      memcpy(&ctx->buffer[used], data, size);
      return;
    }
 
    memcpy(&ctx->buffer[used], data, available);
    data = (const unsigned char *)data + available;
    size -= available;
    body(ctx, ctx->buffer, 64);
  }
 
  if (size >= 64) {
    data = body(ctx, data, size & ~(unsigned long)0x3f);
    size &= 0x3f;
  }
 
  memcpy(ctx->buffer, data, size);
}
 
#define OUT(dst, src)                           \
  (dst)[0] = (unsigned char)(src);              \
  (dst)[1] = (unsigned char)((src) >> 8);       \
  (dst)[2] = (unsigned char)((src) >> 16);      \
  (dst)[3] = (unsigned char)((src) >> 24);
 
__device__ void GPU_MD5_Final(unsigned char *result, GPU_MD5_CTX *ctx) {
  unsigned long used, available;
  
  used = ctx->lo & 0x3f;
  
  ctx->buffer[used++] = 0x80;
  
  available = 64 - used;
  
  if (available < 8) {
    memset(&ctx->buffer[used], 0, available);
    body(ctx, ctx->buffer, 64);
    used = 0;
    available = 64;
  }
  
  memset(&ctx->buffer[used], 0, available - 8);
  
  ctx->lo <<= 3;
  OUT(&ctx->buffer[56], ctx->lo)
    OUT(&ctx->buffer[60], ctx->hi)
 
    body(ctx, ctx->buffer, 64);
 
  OUT(&result[0], ctx->a)
    OUT(&result[4], ctx->b)
    OUT(&result[8], ctx->c)
    OUT(&result[12], ctx->d)
 
    memset(ctx, 0, sizeof(*ctx));
}

__device__ char computeChar(int i) {
  i = i % NUM_CHAR;
  if (i < 26) {
     return (char) i + 97;
} else {
     return (char) i + 22;
  }
}


__global__ void computeMD5(uint8_t* passwordHash, bool* checker, int offset) {
  //printf("made it into computeMD5");
  int new_block_id = blockIdx.x + offset;
  
  char password[LENGTH+1];
  password[7] = computeChar(threadIdx.x);
  password[6] = computeChar(new_block_id*2 + (threadIdx.x / NUM_CHAR));
  password[5] = computeChar(new_block_id / 18);
  password[4] = computeChar(new_block_id / 648);
  password[3] = computeChar(new_block_id / 23328);
  password[2] = computeChar(new_block_id / 839808);
  password[1] = computeChar(new_block_id / 30233088);
  password[0] = computeChar(new_block_id / 108839168);
  password[8] = '\0';
  /*
  if (threadIdx.x == 0) {
  printf("%s\n",password);
  }*/
  
  //Initialize the MD5 context
  GPU_MD5_CTX context;
  GPU_MD5_Init(&context);

  //add our data to MD5
  GPU_MD5_Update(&context, password, LENGTH);

  //Finish
  uint8_t output[MD5_DIGEST_LENGTH];
  GPU_MD5_Final(output, &context);

  //printf("finished computing the hash");
  int match = 0;
  
  for(size_t i=0; i < MD5_DIGEST_LENGTH; i++) {
    if (output[i] == passwordHash[i]) {
      match++;
    }
  }

  if (match == MD5_DIGEST_LENGTH) {
    *checker = true;
    printf("Password has been found on the GPU. It is %s \n", password);
  }
}

typedef struct password_entry {
  char pwd[LENGTH+1];
  uint8_t password_md5[MD5_DIGEST_LENGTH];
} password_entry_t;

/**
 * Read a file of username and MD5 passwords. Return a linked list
 * of entries.
 * \param filename  The path to the password file
 * \returns         A pointer to the first node in the password list
 */
password_entry* read_password_file(const char* filename, int *size) {
  // Open the password file
  FILE* password_file = fopen(filename, "r");
  if (password_file == NULL) {
    perror("opening password file");
    exit(2);
  }

  char length[LENGTH];
  // Get the first line containing the number of passwords
    if(fscanf(password_file, " %s ", length) != 1) {
      fprintf(stderr, "Error reading password file: malformed line\n");
      exit(2);
      }

  *size = atoi(length);

  password_entry* passwords = (password_entry*) malloc(sizeof(password_entry) * *size);
  int i = -1;

  // Read until we hit the end of the file
  while (!feof(password_file) && i < *size) {
  i++;
  
    // Make space to hold the popular password unhashed
    char * passwd = (char *) malloc(sizeof(char) * 9);
    uint8_t * md5_string = (uint8_t *) malloc(sizeof(uint8_t) * MD5_DIGEST_LENGTH * 2 + 1);
    
    // Try to read. The space in the format string is required to eat the newline
    if(fscanf(password_file, " %s ", passwd) != 1) {
      fprintf(stderr, "Error reading password file: malformed line\n");
      exit(2);
    }
   
    // Convert the passwd to a MD5 and store it
    MD5((unsigned char*) passwd, LENGTH,  md5_string);
    
    // Add the new node to the front of the list
    strcpy(passwords[i].pwd, passwd);
    memcpy(passwords[i].password_md5, md5_string, MD5_DIGEST_LENGTH);
  }

  return passwords;
}


__global__ void popularPasswords(uint8_t* passwordHash, password_entry* passwordEntries, bool* checker) {

   int index = (blockIdx.x * THREADS_PER_BLOCK) + threadIdx.x;
   uint8_t* passwordEntry = passwordEntries[index].password_md5;
   
   int match = 0;
  
  for(size_t i=0; i < MD5_DIGEST_LENGTH; i++) {
    if (passwordHash[i] == passwordEntry[i]) {
      match++;
    }
  }

  if (match == MD5_DIGEST_LENGTH) {
    *checker = true;
    printf("Password has been found on the GPU. It is %s \n", passwordEntries[index].pwd);
  }

}


int main() {
  char password[] = "abrakada";
  uint8_t passwordHash[MD5_DIGEST_LENGTH+1];
  bool* checker = (bool*)malloc(sizeof(bool));
  *checker = false;
  int size = 0;
//  char passwordFile[MD5_DIGEST_LENGTH];


    char* filename = "/home/nashgemm/CSC213/213-project/popularpwds";

    printf("Enter in your test password: ");
   scanf("%s", &password);

  // printf("Enter password file: ");
  // scanf("%s", &passwordFile);

   password_entry* passwordEntries = read_password_file(filename, &size);

MD5((unsigned char*) password, LENGTH, passwordHash);
  
  for(size_t i=0; i < MD5_DIGEST_LENGTH; i++) {
   // printf("%x", passwordHash[i]);
  }
  
  uint8_t* gpu_passwordHash;
  bool* gpu_checker;
  //int gpu_size;
  password_entry* gpu_passwordEntries;

  if(hipMalloc(&gpu_passwordEntries, sizeof(password_entry)* size) != hipSuccess) {
    fprintf(stderr, "Failed to allocate memory for passwordEntries\n");
    exit(2);
  }
  /*
  if(hipMalloc(&gpu_size, sizeof(int)) != hipSuccess) {
    fprintf(stderr, "Failed to allocate memory for passwordEntries Size\n");
    exit(2);
  }
  if(hipMemcpy(&gpu_size, &size, sizeof(int),  hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy size to the GPU\n");
  }

*/
  if(hipMemcpy(gpu_passwordEntries, passwordEntries, sizeof(password_entry) * size,  hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy passwordEntries to the GPU\n");
  }
  

  if(hipMalloc(&gpu_passwordHash, (sizeof(uint8_t)* MD5_DIGEST_LENGTH) +1) != hipSuccess) {
    fprintf(stderr, "Failed to allocate memory for passwordHash\n");
    exit(2);
  }
  
  if(hipMalloc(&gpu_checker, sizeof(bool)) != hipSuccess) {
    fprintf(stderr, "Failed to allocate memory for checker\n");
    exit(2);
  }
  
  if(hipMemcpy(gpu_passwordHash, passwordHash, (sizeof(uint8_t) * MD5_DIGEST_LENGTH) + 1,  hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy testHash to the GPU\n");
  }
  
  if(hipMemcpy(gpu_checker, checker, sizeof(bool),  hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy checker to the GPU\n");
  }
  
  
/* EDIT OUT PRINTING
printf("called MD5 in CPU");
  printf("\n%u\n", NUM_BLOCKS);
  printf("\n%u\n", THREADS_PER_BLOCK);
*/

  size_t NUM_BLOCKS = pow(NUM_CHAR, LENGTH)/THREADS_PER_BLOCK;

  popularPasswords<<<1, THREADS_PER_BLOCK>>>(gpu_passwordHash, gpu_passwordEntries, gpu_checker);

  // // COPY STUFF BACK FROM THE GPU
  if(hipMemcpy(checker, gpu_checker, sizeof(bool),  hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy checker from the GPU\n");
  }
  
  if (*checker == true) {
     printf("We found the password on the GPU from the popular file\n");
  } else {

  int i = 0;
  for(; i < 783641; i++) {
    computeMD5<<<50000,THREADS_PER_BLOCK>>>(gpu_passwordHash, gpu_checker, i*50000);
    if(hipMemcpy(checker, gpu_checker, sizeof(bool),  hipMemcpyDeviceToHost) != hipSuccess) {
     fprintf(stderr, "Failed to copy checker from the GPU\n");
    }
    if (*checker) {
      break;
    }
    if(hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "the error came from inside the kernel...comes back\n");
    fprintf(stderr, "%s\n", hipGetErrorString(hipPeekAtLastError()));
    }
  }
  if (!(*checker)) {
  computeMD5<<<32048, THREADS_PER_BLOCK>>>(gpu_passwordHash, gpu_checker, i*50000);
  }
  
  if(hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "the error came from inside the kernel...comes back\n");
    fprintf(stderr, "%s\n", hipGetErrorString(hipPeekAtLastError()));
  }
  
  // // COPY STUFF BACK FROM THE GPU
  if(hipMemcpy(checker, gpu_checker, sizeof(bool),  hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy checker from the GPU\n");
  }
  
  if (*checker == true) {
     printf("We found the password on the GPU the brute force time \n");
     // Add the password to the list.
     // Timings
       FILE* password_file = fopen(filename, "a");
  if (password_file == NULL) {
    perror("opening password file");
    exit(2);
  }
        fprintf(password_file, "%s\n", password);
        fclose(password_file);
  /*      
  FILE* password_file_size = fopen(filename, "w");
  if (password_file == NULL) {
    perror("opening password file");
    exit(2);
  }
  fprintf(password_file_size, "%s\n", size+1);
  fclose(password_file_size);
    */   
  }
}
  hipFree(gpu_passwordEntries);
  //hipFree(gpu_size);
  hipFree(gpu_checker);
  hipFree(gpu_passwordHash);
  return 0;
}
